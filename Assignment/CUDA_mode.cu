#include "hip/hip_runtime.h"
#include "CUDA_mode.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <string>
#include <hip/hip_runtime.h>
#include <>
#include "Common.h"  // For the global isPrint flag

using namespace std;

// Constructor: Initialize the SimplexCUDA object with matrix A, vector B, and vector C
SimplexCUDA::SimplexCUDA(vector<vector<float>> matrix, vector<float> b, vector<float> c) {
    // Flatten matrix A into a single 1D array
    A_flat = vector<float>(matrix.size() * matrix[0].size());
    for (int i = 0; i < matrix.size(); i++) {
        for (int j = 0; j < matrix[0].size(); j++) {
            A_flat[i * matrix[0].size() + j] = matrix[i][j];  // Flatten A into 1D array
        }
    }

    A = matrix;
    B = b;
    C = c;
    rows = matrix.size();
    cols = matrix[0].size();
    maximum = 0;

    // Allocate device memory
    hipMalloc(&d_A, rows * cols * sizeof(float));
    hipMalloc(&d_B, rows * sizeof(float));
    hipMalloc(&d_C, cols * sizeof(float));
    hipMalloc(&d_maximum, sizeof(float));
    hipMalloc(&d_isOptimal, sizeof(bool));
    hipMalloc(&d_pivotColumn, sizeof(int));
    hipMalloc(&d_pivotRow, sizeof(int));

    // Copy data to device
    hipMemcpy(d_A, A_flat.data(), rows * cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.data(), rows * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C.data(), cols * sizeof(float), hipMemcpyHostToDevice);

    float h_maximum = 0;
    hipMemcpy(d_maximum, &h_maximum, sizeof(float), hipMemcpyHostToDevice);
}

// Device function to find the pivot column
__device__ void findPivotColumnDevice(float* C, int cols, bool* isOptimal, int* pivotColumn) {
    int tid = threadIdx.x;
    __shared__ float minValue;
    __shared__ int minIndex;

    if (tid == 0) {
        minValue = C[0];
        minIndex = 0;
        for (int i = 1; i < cols; i++) {
            if (C[i] < minValue) {
                minValue = C[i];
                minIndex = i;
            }
        }
        *isOptimal = (minValue >= 0);
        *pivotColumn = minIndex;
    }
    __syncthreads();
}

// Device function to find the pivot row
__device__ void findPivotRowDevice(float* A, float* B, int rows, int cols, int pivotColumn, int* pivotRow) {
    int tid = threadIdx.x;
    __shared__ float minRatio;
    __shared__ int minIndex;

    if (tid == 0) {
        minRatio = FLT_MAX;
        minIndex = -1;
        for (int i = 0; i < rows; i++) {
            float a = A[i * cols + pivotColumn];
            if (a > 0) {
                float ratio = B[i] / a;
                if (ratio < minRatio) {
                    minRatio = ratio;
                    minIndex = i;
                }
            }
        }
        *pivotRow = minIndex;
    }
    __syncthreads();
}

// Kernel to perform one iteration of the simplex method
__global__ void simplexIteration(float* A, float* B, float* C, int rows, int cols, float* maximum, bool* isOptimal, int* pivotColumn, int* pivotRow) {
    if (threadIdx.x == 0) {
        // Initialize isOptimal flag
        *isOptimal = true;
    }
    __syncthreads();

    // Find pivot column
    findPivotColumnDevice(C, cols, isOptimal, pivotColumn);
    __syncthreads();

    if (*isOptimal) {
        return;  // Optimal solution found
    }

    // Find pivot row
    findPivotRowDevice(A, B, rows, cols, *pivotColumn, pivotRow);
    __syncthreads();

    // Get pivot value
    float pivotValue = A[*pivotRow * cols + *pivotColumn];

    // Normalize pivot row
    for (int j = threadIdx.x; j < cols; j += blockDim.x) {
        A[*pivotRow * cols + j] /= pivotValue;
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        B[*pivotRow] /= pivotValue;
    }
    __syncthreads();

    // Update other rows
    for (int i = 0; i < rows; i++) {
        if (i != *pivotRow) {
            float factor = A[i * cols + *pivotColumn];
            for (int j = threadIdx.x; j < cols; j += blockDim.x) {
                A[i * cols + j] -= factor * A[*pivotRow * cols + j];
            }
            __syncthreads();
            if (threadIdx.x == 0) {
                B[i] -= factor * B[*pivotRow];
            }
            __syncthreads();
        }
    }

    // Update objective function
    float factor = C[*pivotColumn];
    for (int j = threadIdx.x; j < cols; j += blockDim.x) {
        C[j] -= factor * A[*pivotRow * cols + j];
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        *maximum += factor * B[*pivotRow];
    }
    __syncthreads();
}

// Calculate the Simplex solution
vector<float> SimplexCUDA::CalculateSimplex() {
    bool h_isOptimal = false;

    // Loop until optimality is reached
    while (!h_isOptimal) {
        // Launch the simplex iteration kernel
        simplexIteration << <1, 1024 >> > (d_A, d_B, d_C, rows, cols, d_maximum, d_isOptimal, d_pivotColumn, d_pivotRow);
        hipDeviceSynchronize();

        // Copy optimality flag back to host
        hipMemcpy(&h_isOptimal, d_isOptimal, sizeof(bool), hipMemcpyDeviceToHost);
    }

    // Copy results back to host
    hipMemcpy(A_flat.data(), d_A, rows * cols * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(B.data(), d_B, rows * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(C.data(), d_C, cols * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&maximum, d_maximum, sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_maximum);
    hipFree(d_isOptimal);
    hipFree(d_pivotColumn);
    hipFree(d_pivotRow);

    printResults("maximization");

    return B; // Return the solution (values of the variables)
}

// Print the results
void SimplexCUDA::printResults(const string& mode) {
    if (isPrint) {
        cout << fixed << setprecision(2);
        cout << YELLOW << "Solution for the variables: " << RESET << endl;
        int numRows = A.size();
        int numCols = A[0].size();
        for (int i = 0; i < numRows; i++) {
            for (int j = 0; j < numCols; j++) {
                A[i][j] = A_flat[i * numCols + j];  // Convert 1D array back to 2D
            }
        }
        for (int i = 0; i < A.size(); i++)
        { // every basic column has the values, get it form B array
            int count0 = 0;
            int index = 0;
            for (int j = 0; j < rows; j++)
            {
                if (A[j][i] == 0.0)
                {
                    count0 += 1;
                }
                else if (A[j][i] == 1)
                {
                    index = j;
                }
            }

            if (count0 == rows - 1)
            {
                cout << GREEN << "Variable " << i + 1 << ": " << BOLD << B[index] << RESET << endl;
            }
            else
            {
                cout << GREEN << "Variable " << i + 1 << ": " << BOLD << 0 << RESET << endl;
            }
        }
    }
    cout << YELLOW << "The " << mode << " value of the objective function is: " << BOLD << maximum << RESET << endl;
}
